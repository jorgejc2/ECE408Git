#include "hip/hip_runtime.h"
// MP 1
#include <wb.h>

__global__ void vecAdd(float *in1, float *in2, float *out, int len) {
  //@@ Insert code to implement vector addition here
  int i = blockIdx.x * blockDim.x + threadIdx.x;

  if (i<len) 
    out[i] = in1[i] + in2[i];
}

int main(int argc, char **argv) {
  wbArg_t args;
  int inputLength; // size of array
  float *hostInput1; // operand 1 on cpu
  float *hostInput2; // operand 2 on cpu
  float *hostOutput; // output on cpu
  float *deviceInput1; // operand 1 on gpu
  float *deviceInput2; // operand 2 on gpu
  float *deviceOutput; // output on gpu

  args = wbArg_read(argc, argv);

  wbTime_start(Generic, "Importing data and creating memory on host");
  hostInput1 =
      (float *)wbImport(wbArg_getInputFile(args, 0), &inputLength);
  hostInput2 =
      (float *)wbImport(wbArg_getInputFile(args, 1), &inputLength);
  hostOutput = (float *)malloc(inputLength * sizeof(float));
  wbTime_stop(Generic, "Importing data and creating memory on host");

  wbLog(TRACE, "The input length is ", inputLength);

  wbTime_start(GPU, "Allocating GPU memory.");
  //@@ Allocate GPU memory here
  int size = inputLength * sizeof(float);

  /* allocate memory for first input */
  hipMalloc((void **) &deviceInput1, size);

  /* allocate memory for second input */
  hipMalloc((void **) &deviceInput2, size);

  /* allocate memory for output */
  hipMalloc((void **) &deviceOutput, size);

  wbTime_stop(GPU, "Allocating GPU memory.");

  wbTime_start(GPU, "Copying input memory to the GPU.");
  //@@ Copy memory to the GPU here

  /* copy first input */
  hipMemcpy(deviceInput1, hostInput1, size, hipMemcpyHostToDevice);

  /* copy second input */
  hipMemcpy(deviceInput2, hostInput2, size, hipMemcpyHostToDevice);

  wbTime_stop(GPU, "Copying input memory to the GPU.");

  //@@ Initialize the grid and block dimensions here
  dim3 DimGrid(ceil(inputLength/256.0), 1, 1);
  dim3 DimBlock(256,1,1);

  wbTime_start(Compute, "Performing CUDA computation");
  //@@ Launch the GPU Kernel here
  vecAdd<<<DimGrid, DimBlock>>>(deviceInput1, deviceInput2, deviceOutput, inputLength);

  hipDeviceSynchronize();
  wbTime_stop(Compute, "Performing CUDA computation");

  wbTime_start(Copy, "Copying output memory to the CPU");
  //@@ Copy the GPU memory back to the CPU here
  hipMemcpy(hostOutput, deviceOutput, size, hipMemcpyDeviceToHost);

  wbTime_stop(Copy, "Copying output memory to the CPU");

  wbTime_start(GPU, "Freeing GPU Memory");
  //@@ Free the GPU memory here
  hipFree(deviceInput1);
  hipFree(deviceInput2);
  hipFree(deviceOutput);


  wbTime_stop(GPU, "Freeing GPU Memory");

  wbSolution(args, hostOutput, inputLength);

  free(hostInput1);
  free(hostInput2);
  free(hostOutput);

  return 0;
}
