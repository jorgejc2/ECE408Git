#include "hip/hip_runtime.h"
#include <cmath>
#include <iostream>
#include "gpu-new-forward.h"

#define TILE_WIDTH 16
#define UNROLL_BLOCK_SIZE 256
#define BLOCK_SIZE 1024

/*
** THIS IS FOR THE FINAL SUBMISSION OF OPTIMES LESS THAN 70 MS
**
*/

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

__constant__ float mask_kernel[3136];

static int max_threads_per_block;
static int max_shared_size;
static int warp_size;

__global__ void conv_forward_kernel_shared_mem(float *output, const float *input, const float *mask, const int Batch, const int Map_out, const int Channel, const int Height, const int Width, const int K)
{
    /*
    Modify this function to implement the forward pass described in Chapter 16.
    We have added an additional dimension to the tensors to support an entire mini-batch
    The goal here is to be correct AND fast.

    Function paramter definitions:
    output - output
    input - input
    mask - convolution kernel
    Batch - batch_size (number of images in x)
    Map_out - number of output feature maps
    Channel - number of input feature maps
    Height - input height dimension
    Width - input width dimension
    K - kernel height and width (K x K)
    */

    int tile_width = blockDim.x;

    const int Height_out = Height - K + 1;
    const int Width_out = Width - K + 1;
    const int W_grid = ceil((float)Width_out / tile_width);
    const int H_grid = ceil((float)Height_out / tile_width);

    // We have some nice #defs for you below to simplify indexing. Feel free to use them, or create your own.
    // An example use of these macros:
    // float a = in_4d(0,0,0,0)
    // out_4d(0,0,0,0) = a

    #define out_4d(i3, i2, i1, i0) output[(i3) * (Map_out * Height_out * Width_out) + (i2) * (Height_out * Width_out) + (i1) * (Width_out) + i0]
    #define in_4d(i3, i2, i1, i0) input[(i3) * (Channel * Height * Width) + (i2) * (Height * Width) + (i1) * (Width) + i0]
    #define mask_4d(i3, i2, i1, i0) mask[(i3) * (Channel * K * K) + (i2) * (K * K) + (i1) * (K) + i0]
    #define const_mask_4d(i3, i2, i1, i0) mask_kernel[(i3) * (Channel * K * K) + (i2) * (K * K) + (i1) * (K) + i0]

    // Insert your GPU convolution kernel code here

    int n, m , h0, w0, h_base, w_base, h, w;
    int X_tile_width = tile_width + K - 1;
    extern __shared__ float shmem[];
    float* X_shared = &shmem[0];
    float* W_shared = &shmem[X_tile_width * X_tile_width];
    n = blockIdx.x; // current batch sample
    m = blockIdx.y; // current map output feature
    h0 = threadIdx.y; 
    w0 = threadIdx.x;
    h_base = (blockIdx.z/W_grid)*tile_width; // vertical base out data index for the block
    w_base = (blockIdx.z % W_grid)*tile_width; // horizontal base out data index for the block 
    h = h_base + h0;
    w = w_base + w0;

    #define x_share(i1, i0) X_shared[(i1)*(X_tile_width) + (i0)]
    #define w_share(i1, i0) W_shared[(i1)*(K) + (i0)]

    float acc = 0; // initialize output result

    /* iterate through all the channels */
    for (int c = 0; c < Channel; c++) {
        if ((h0 < K) && (w0 < K)) {
            // w_share(h0, w0) = mask_4d(m, c, h0, w0);
            w_share(h0, w0) = const_mask_4d(m, c, h0, w0);
        }
        // else
        //     W_shared[h0, w0] = 0;
        // __syncthreads();

        for (int i = h; i < h_base + X_tile_width; i += tile_width) {
            for (int j = w; j < w_base + X_tile_width; j += tile_width) {
                if ((i < Height) && (j < Width)) {
                    x_share(i - h_base, j - w_base) = in_4d(n, c, i, j);
                }
                else {
                    x_share(i - h_base, j - w_base) = 0;
                }
            }
        }
        __syncthreads();

        for (int p = 0; p < K; p++) {
            for (int q = 0; q < K; q++) {
                acc += x_share(h0 + p, w0 + q) * w_share(p, q);
            }
        }
        __syncthreads();
    }
    if ((h < Height_out) && (w < Width_out)) {
        out_4d(n, m, h, w) = acc;
    }
    
    #undef out_4d
    #undef in_4d
    #undef mask_4d
    #undef x_share
    #undef w_share
    #undef const_mask_4d
}

/* code for register tiling of matrix multiplication */
#define TILE_SZ_A 32
#define TILE_SZ_B 16
#define TILE_SZ_RATIO (TILE_SZ_A / TILE_SZ_B)

__global__ void mygemm(float * __restrict__ c, //<! [out] and MxN matrix
                       const float *a,        //<! [in] an MxK matrix
                       const float *b,        //<! [in] an KxN matrix
                       const int M, const int K, const int N, const int K_in, const int Channel, int H, int W) {

// Macros for accessing flattened matrices
#define A(i1, i0) a[(i1) * K + (i0)] // this will be the mask
#define B(i2, i1, i0) b[(i2) * (K*N) + (i1)*N + (i0)]
#define C(i2, i1, i0) c[(i2)*(M*N) + (i1)*N + (i0)]
// #define A(i1, i0) mask_kernel[(i1) * K + (i0)]

#define in_4d(i3, i2, i1, i0) b[(i3) * (Channel * H * W) + (i2) * (H * W) + (i1) * (W) + i0]

// if ((threadIdx.x + threadIdx.y + threadIdx.z == 0) && (blockIdx.x + blockIdx.y + blockIdx.z == 0))
        // printf("Gemm kernel runs");

int curr_batch = blockIdx.z;

  // Shared memory for tiling input B array
  __shared__ float B_s[TILE_SZ_RATIO][TILE_SZ_B];

  // Index variables
  const unsigned int row = blockDim.x * blockIdx.x + threadIdx.x;
  const unsigned int col = blockIdx.y * TILE_SZ_B;

  int W_out = W - K_in + 1;
  int H_out = H - K_in + 1;

  // Privatization of output variables
  float c_reg[TILE_SZ_B];

  // Initialize output values
  for (unsigned int outIdx = 0; outIdx < TILE_SZ_B; ++outIdx) {
    c_reg[outIdx] = 0;
  }

  const unsigned int i = threadIdx.x / TILE_SZ_B;
  const unsigned int j = threadIdx.x % TILE_SZ_B;

  // Loop over the input tiles
  for (unsigned int tileIdx = 0; tileIdx < ceil(K/(1.0 * TILE_SZ_RATIO)); ++tileIdx) {
    // Load the tile of B into shared memory
    if (tileIdx * TILE_SZ_RATIO + i < K && col + j < N) {
        int curr_channel = (tileIdx * TILE_SZ_RATIO + i) / (K_in*K_in);
        int m = (col + j) % W_out;
        int n = (col + j) / W_out;
        int p = ((tileIdx * TILE_SZ_RATIO + i) - (curr_channel*K_in*K_in)) % K_in;
        int q = ((tileIdx * TILE_SZ_RATIO + i) - (curr_channel*K_in*K_in)) / K_in;
        int x = m + p;
        int y = n + q;
        //B_s[i][j] = B(curr_batch, tileIdx * TILE_SZ_RATIO + i, col + j);
        B_s[i][j] = in_4d(curr_batch, curr_channel, y, x);
    } else {
      B_s[i][j] = 0;
    }
    __syncthreads();
    // Loop over elements inside the tile
    for (unsigned int idx = 0; idx < TILE_SZ_RATIO; ++idx) {
      // Load tile of A matrix into register
      float a_reg;
      if (row < M && tileIdx * TILE_SZ_RATIO + idx < K) {
        a_reg = A(row, tileIdx * TILE_SZ_RATIO + idx);
      } else {
        a_reg  = 0;
      }
      // Loop over and update the output elemena_regts assigned to the thread
      for (unsigned int outIdx = 0; outIdx < TILE_SZ_B; ++outIdx) {
        c_reg[outIdx] += a_reg * B_s[idx][outIdx];
      }
    }
    __syncthreads();
  }

  for (unsigned int outIdx = 0; outIdx < TILE_SZ_B; ++outIdx) {
    if (row < M && col + outIdx < N) {
      C(curr_batch, row, col + outIdx) = c_reg[outIdx];
    }
  }

#undef A
#undef in_4d
#undef B
#undef C
}

__host__ void GPUInterface::conv_forward_gpu_prolog(const float *host_output, const float *host_input, const float *host_mask, float **device_output_ptr, float **device_input_ptr, float **device_mask_ptr, const int Batch, const int Map_out, const int Channel, const int Height, const int Width, const int K)
{
    // Allocate memory and copy over the relevant data structures to the GPU

    // We pass double pointers for you to initialize the relevant device pointers,
    //  which are passed to the other two functions.

    // Useful snippet for error checking
    // hipError_t error = hipGetLastError();
    // if(error != hipSuccess)
    // {
    //     std::cout<<"CUDA error: "<<hipGetErrorString(error)<<std::endl;
    //     exit(-1);
    // }
    // get_device_properties(); // print and save some of the properties of the gpu
    const int Height_out = Height - K + 1;
    const int Width_out = Width - K + 1;
    int inputSize = Batch*Channel*Height*Width*sizeof(float);
    int outputSize = Batch*Map_out*Height_out*Width_out*sizeof(float);
    int maskSize = Map_out*Channel*K*K*sizeof(float);
    hipMalloc((void **)device_input_ptr, inputSize);
    hipMalloc((void **)device_output_ptr, outputSize);

    /* optimization that places mask into constant memory */
    int kernel_size = sizeof(float) * Map_out * Channel * K * K;

    hipMemcpy(*device_input_ptr, host_input, inputSize, hipMemcpyHostToDevice);

    if(Channel == 1){
      hipMemcpyToSymbol(HIP_SYMBOL(mask_kernel), host_mask, kernel_size);
    }else{
      hipMalloc((void **)device_mask_ptr, maskSize);
      hipMemcpy(*device_mask_ptr, host_mask, maskSize, hipMemcpyHostToDevice);
    }
}


__host__ void GPUInterface::conv_forward_gpu(float *device_output, const float *device_input, const float *device_mask, const int Batch, const int Map_out, const int Channel, const int Height, const int Width, const int K)
{
    int W_out = Width - K + 1;
    int H_out = Height - K + 1;
    int H_unroll = Channel * K * K;
    int W_unroll = H_out * W_out;


    if(Channel == 1){
      get_device_properties();
      int optimal_tile_width = sqrt(max_threads_per_block);
      int num_blocks = -1;

      optimal_tile_width = 16;
      printf("Optimal tile width is %d", optimal_tile_width);
      // Set the kernel dimensions and call the kernel
      int N = Batch; // blockDim.x will corresponds to the batch sample
      int M = Map_out; // blockDim.y corresponds to the output feature
      int W_grid = ceil((float)W_out / optimal_tile_width);
      int H_grid = ceil((float)H_out / optimal_tile_width);
      int Z = W_grid * H_grid; // the current output tile to be computed in the output feature

      dim3 blockDim(optimal_tile_width, optimal_tile_width, 1);
      dim3 gridDim(N, M, Z);

      size_t shmem_size = sizeof(float) * ((optimal_tile_width + K - 1) * (optimal_tile_width + K - 1) + K*K);
      conv_forward_kernel_shared_mem<<<gridDim, blockDim, shmem_size>>>(device_output, device_input, device_mask, Batch, Map_out, Channel, Height, Width, K);

    }else{
      dim3 dimGrid(ceil(Map_out / (1.0*TILE_SZ_A)), ceil(W_unroll / (1.0*TILE_SZ_B)), Batch);
      dim3 dimBlock(TILE_SZ_A, 1, 1);
      mygemm<<<dimGrid, dimBlock>>>(device_output, device_mask, device_input, Map_out, Channel*K*K, H_out*W_out, K, Channel, Height, Width);
    }
    // gpuErrchk( hipPeekAtLastError() );
    // gpuErrchk( hipDeviceSynchronize() );
}


__host__ void GPUInterface::conv_forward_gpu_epilog(float *host_output, float *device_output, float *device_input, float *device_mask, const int Batch, const int Map_out, const int Channel, const int Height, const int Width, const int K)
{
    const int Height_out = Height - K + 1;
    const int Width_out = Width - K + 1;
    int outputSize = Batch*Map_out*Height_out*Width_out*sizeof(float);

    // Copy the output back to host
    hipMemcpy(host_output, device_output, outputSize, hipMemcpyDeviceToHost);    

    // Free device memory
    hipFree(device_output);
    hipFree(device_input);

    if(Channel == 4) hipFree(device_mask);
}

__host__ void GPUInterface::get_device_properties()
{
    int deviceCount;
    hipGetDeviceCount(&deviceCount);

    for(int dev = 0; dev < deviceCount; dev++)
    {
        hipDeviceProp_t deviceProp;
        hipGetDeviceProperties(&deviceProp, dev);

        std::cout<<"Device "<<dev<<" name: "<<deviceProp.name<<std::endl;
        std::cout<<"Computational capabilities: "<<deviceProp.major<<"."<<deviceProp.minor<<std::endl;
        std::cout<<"Max Global memory size: "<<deviceProp.totalGlobalMem<<std::endl;
        std::cout<<"Max Constant memory size: "<<deviceProp.totalConstMem<<std::endl;
        std::cout<<"Max Shared memory size per block: "<<deviceProp.sharedMemPerBlock<<std::endl;
        max_shared_size = deviceProp.sharedMemPerBlock;
        std::cout<<"Max threads per block: "<<deviceProp.maxThreadsPerBlock<<std::endl;
        max_threads_per_block = deviceProp.maxThreadsPerBlock;
        std::cout<<"Max block dimensions: "<<deviceProp.maxThreadsDim[0]<<" x, "<<deviceProp.maxThreadsDim[1]<<" y, "<<deviceProp.maxThreadsDim[2]<<" z"<<std::endl;
        std::cout<<"Max grid dimensions: "<<deviceProp.maxGridSize[0]<<" x, "<<deviceProp.maxGridSize[1]<<" y, "<<deviceProp.maxGridSize[2]<<" z"<<std::endl;
        std::cout<<"Warp Size: "<<deviceProp.warpSize<<std::endl;
        warp_size = deviceProp.warpSize;
    }
}
