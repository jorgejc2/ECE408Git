#include "hip/hip_runtime.h"
#include <cmath>
#include <iostream>
#include "gpu-new-forward.h"

#define TILE_WIDTH 16
#define UNROLL_BLOCK_SIZE 256
#define BLOCK_SIZE 1024

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

static int max_threads_per_block;
static int max_shared_size;
static int warp_size;

__constant__ float mask_kernel[3136];

__global__ void unroll_Kernel (int C, int H, int W, int K, const float* X, float* X_unroll) {
    // C = number of channels which should always be 3
    // H = height of input images
    // W = width of input images
    // n = current batch sample
    // X = array to a batch's input images, an image for each channel (so 3 images)
    // X_unroll = output where the input images will get mapped to
    // if ((threadIdx.x + threadIdx.y + threadIdx.z == 0) && (blockIdx.x + blockIdx.y + blockIdx.z == 0))
        // printf("Unroll kernel runs");
    int c, s, h_out, w_out, h_unroll, w_unroll, w_base, p, q;
    int t = blockIdx.x * UNROLL_BLOCK_SIZE + threadIdx.x;
    int H_out = H - K + 1;
    int W_out = W - K + 1;
    int W_unroll = H_out * W_out;
    int H_unroll = C*K*K;

    int n = blockIdx.y; // the current batch sample

    // #define X_unroll_output(i1, i0) X_unroll[(i1) * (C * K * K) + (i0)]
    #define X_unroll_output(i2, i1, i0) X_unroll[(i2) * (W_out*H_out*C*K*K) + (i1) * (W_out * H_out) + (i0)]
    // #define X_input(i2, i1, i0) X[(i2) * (C * W) + (i1) * W + (i0)]
    #define in_4d(i3, i2, i1, i0) X[(i3) * (C * H * W) + (i2) * (H * W) + (i1) * (W) + i0]

    if (t < C * W_unroll) {
        c = t / W_unroll; // channel based on index
        s = t % W_unroll; // section currently working with 
        // c = t / H_unroll; // channel based on index
        // s = t % H_unroll; // section currently working with 
        h_out = s / W_out;
        w_out = s % W_out;
        // h_unroll = h_out * W_out + w_out;
        w_unroll = h_out * W_out + w_out;
        w_base = c * K * K;

        for (p = 0; p < K; p++) {
            for (q = 0; q < K; q++) {
                // w_unroll = w_base + p*K + q;
                h_unroll = w_base + p*K + q;
                // X_output(h_unroll, w_unroll) = X_input(c, h_out + p, w_out + q);
                X_unroll_output(n, h_unroll, w_unroll) = in_4d(n, c, h_out + p, w_out + q);
            }
        }
    }

    #undef X_unroll_output
    #undef in_4d
}

// Compute C = A * B
__global__ void matrixMultiply(const float* A, const float* B, float* __restrict__ C, int numARows,
                               int numAColumns, int numBRows,
                               int numBColumns, int numCRows,
                               int numCColumns) {
  //@@ Insert code to implement matrix multiplication here

  int tx = threadIdx.x;
  int ty = threadIdx.y;
  int bx = blockIdx.x;
  int by = blockIdx.y;

  int currBatch = blockIdx.z;

  /* used in my tile multiplication */
  __shared__ float Mds[TILE_WIDTH][TILE_WIDTH];
  __shared__ float Nds[TILE_WIDTH][TILE_WIDTH];

  /* identify the column and row of the current thread */
  int row = by * TILE_WIDTH + ty;  //position in the full matrix
  int col = bx * TILE_WIDTH + tx;  //position in the full matrix

  /*
  Remember:
  numAColumns = A's width
  numBColumns = B's width
  numAColumns == numBRows
  */

  float Pvalue = 0;

  /* the number of tiles to loop over is equal to aRows or bColumns */
  /* loop over the M and N tiles required to compute P element */
  for (int ph = 0; ph < ceil(1.0*numAColumns/TILE_WIDTH); ++ph) {

    /* collaborative loading of M and N tiles into shared memory */
    if ((row < numARows) && ((ph*TILE_WIDTH + tx) < numAColumns))
      Mds[ty][tx] = A[row*numAColumns + (ph*TILE_WIDTH + tx)];
    else
      Mds[ty][tx] = 0;

    if (((ph*TILE_WIDTH + ty) < numBRows) && (col < numBColumns))
      Nds[ty][tx] = B[currBatch*(numBRows * numBColumns) + (ph*TILE_WIDTH + ty)*numBColumns + col];
    else
      Nds[ty][tx] = 0;
    __syncthreads();

    for (int k = 0; k < TILE_WIDTH; k++)
      Pvalue += Mds[ty][k] * Nds[k][tx];
    __syncthreads();

    
  }

  if ((row < numCRows) && (col < numCColumns))
      C[currBatch*numCRows*numCColumns + row*numCColumns + col] = Pvalue; // need to use current batch to correctly index output
}
	
__host__ void GPUInterface::conv_forward_gpu_prolog(const float *host_output, const float *host_input, const float *host_mask, float **device_output_ptr, float **device_input_ptr, float **device_mask_ptr, const int Batch, const int Map_out, const int Channel, const int Height, const int Width, const int K)
{
    // Allocate memory and copy over the relevant data structures to the GPU

    // We pass double pointers for you to initialize the relevant device pointers,
    //  which are passed to the other two functions.

    // Useful snippet for error checking
    // hipError_t error = hipGetLastError();
    // if(error != hipSuccess)
    // {
    //     std::cout<<"CUDA error: "<<hipGetErrorString(error)<<std::endl;
    //     exit(-1);
    // }
    // get_device_properties(); // print and save some of the properties of the gpu
    const int Height_out = Height - K + 1;
    const int Width_out = Width - K + 1;
    int inputSize = Batch*Channel*Height*Width*sizeof(float);
    int outputSize = Batch*Map_out*Height_out*Width_out*sizeof(float);
    int maskSize = Map_out*Channel*K*K*sizeof(float);
    hipMalloc((void **)device_input_ptr, inputSize);
    hipMalloc((void **)device_mask_ptr, maskSize);
    hipMalloc((void **)device_output_ptr, outputSize);

    /* optimization that places mask into constant memory */
    int kernel_size = sizeof(float) * Map_out * Channel * K * K;
    // hipMemcpyToSymbol(HIP_SYMBOL(mask_kernel), host_mask, kernel_size);

    hipMemcpy(*device_input_ptr, host_input, inputSize, hipMemcpyHostToDevice);
    hipMemcpy(*device_mask_ptr, host_mask, maskSize, hipMemcpyHostToDevice);

}


__host__ void GPUInterface::conv_forward_gpu(float *device_output, const float *device_input, const float *device_mask, const int Batch, const int Map_out, const int Channel, const int Height, const int Width, const int K)
{
    int W_out = Width - K + 1;
    int H_out = Height - K + 1;
    int H_unroll = Channel * K * K;
    int W_unroll = H_out * W_out;
    // float* X_unrolled = (float*)malloc(W_unroll * H_unroll * sizeof(float));
    float* X_unrolled_device;
    
    gpuErrchk(hipMalloc((void **)&X_unrolled_device, sizeof(float)*W_unroll*H_unroll*Batch));

    // printf("Global memory is %lu\n", sizeof(float) * W_unroll * H_unroll * Batch);
    // printf("Gemm: gridx = %lu; gridy = %lu; gridz = %lu\n", (unsigned long)(ceil(Map_out / (1.0*TILE_SZ_A))), (unsigned long)(ceil(W_unroll / (1.0*TILE_SZ_B))), (unsigned long)(Batch));
    // printf("Shared memory size is %d", sizeof(float)*TILE_SZ_RATIO*TILE_SZ_B);

    /* now throw it into the matrix multiplication */

    int num_blocks = ceil((float)(H_out*W_out*Channel) / UNROLL_BLOCK_SIZE);
    dim3 dimUGrid(num_blocks, Batch, 1);
    dim3 dimUBlock(UNROLL_BLOCK_SIZE, 1, 1);
    unroll_Kernel<<<dimUGrid, dimUBlock>>>(Channel, Height, Width, K, device_input, X_unrolled_device);
    // gpuErrchk( hipPeekAtLastError() );
    // gpuErrchk( hipDeviceSynchronize() );

    dim3 dimGrid(ceil((float)W_unroll/TILE_WIDTH), ceil((float)Map_out/TILE_WIDTH), Batch);
    dim3 dimBlock(TILE_WIDTH, TILE_WIDTH, 1);
    matrixMultiply<<<dimGrid, dimBlock>>>(device_mask, X_unrolled_device, device_output, Map_out, Channel*K*K, H_unroll, W_unroll, Map_out, W_unroll);
    gpuErrchk( hipPeekAtLastError() );
    gpuErrchk( hipDeviceSynchronize() );
    hipDeviceSynchronize();

    hipFree(X_unrolled_device);
}


__host__ void GPUInterface::conv_forward_gpu_epilog(float *host_output, float *device_output, float *device_input, float *device_mask, const int Batch, const int Map_out, const int Channel, const int Height, const int Width, const int K)
{
    const int Height_out = Height - K + 1;
    const int Width_out = Width - K + 1;
    int outputSize = Batch*Map_out*Height_out*Width_out*sizeof(float);

    // Copy the output back to host
    hipMemcpy(host_output, device_output, outputSize, hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(device_output);
    hipFree(device_input);
    hipFree(device_mask);

}

__host__ void GPUInterface::get_device_properties()
{
    int deviceCount;
    hipGetDeviceCount(&deviceCount);

    for(int dev = 0; dev < deviceCount; dev++)
    {
        hipDeviceProp_t deviceProp;
        hipGetDeviceProperties(&deviceProp, dev);

        std::cout<<"Device "<<dev<<" name: "<<deviceProp.name<<std::endl;
        std::cout<<"Computational capabilities: "<<deviceProp.major<<"."<<deviceProp.minor<<std::endl;
        std::cout<<"Max Global memory size: "<<deviceProp.totalGlobalMem<<std::endl;
        std::cout<<"Max Constant memory size: "<<deviceProp.totalConstMem<<std::endl;
        std::cout<<"Max Shared memory size per block: "<<deviceProp.sharedMemPerBlock<<std::endl;
        max_shared_size = deviceProp.sharedMemPerBlock;
        std::cout<<"Max threads per block: "<<deviceProp.maxThreadsPerBlock<<std::endl;
        max_threads_per_block = deviceProp.maxThreadsPerBlock;
        std::cout<<"Max block dimensions: "<<deviceProp.maxThreadsDim[0]<<" x, "<<deviceProp.maxThreadsDim[1]<<" y, "<<deviceProp.maxThreadsDim[2]<<" z"<<std::endl;
        std::cout<<"Max grid dimensions: "<<deviceProp.maxGridSize[0]<<" x, "<<deviceProp.maxGridSize[1]<<" y, "<<deviceProp.maxGridSize[2]<<" z"<<std::endl;
        std::cout<<"Warp Size: "<<deviceProp.warpSize<<std::endl;
        warp_size = deviceProp.warpSize;
    }
}
